#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <stdio.h>

__global__ void vectorAdditionKernel(double* A, double* B, double* C, int arraySize) {
	// Get thread ID.
	int threadID = blockDim.x * blockIdx.x + threadIdx.x;

	// Check if thread is within array bounds.
	if (threadID < arraySize) {
		// Add a and b.
		C[threadID] = A[threadID] + B[threadID];
	}
}

/**
* Wrapper function for the CUDA kernel function.
* @param A Array A.
* @param B Array B.
* @param C Sum of array elements A and B directly across.
* @param arraySize Size of arrays A, B, and C.
*/
void kernel(double* A, double* B, double* C, int arraySize) {

	// Initialize device pointers.
	double* d_A, *d_B, *d_C;

	// Allocate device memory.
	hipMalloc((void**)&d_A, arraySize * sizeof(double));
	hipMalloc((void**)&d_B, arraySize * sizeof(double));
	hipMalloc((void**)&d_C, arraySize * sizeof(double));

	// Transfer arrays a and b to device.
	hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);

	// Calculate blocksize and gridsize.
	dim3 blockSize(512, 1, 1);
	dim3 gridSize(512 / arraySize + 1, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	// Launch CUDA kernel.
	vectorAdditionKernel <<<gridSize, blockSize >> > (d_A, d_B, d_C, arraySize);
		
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	printf("Effective Bandwidth (GB/s): %f", 3 * 4 * 3 / milliseconds);

	// Copy result array c back to host memory.
	hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);
}