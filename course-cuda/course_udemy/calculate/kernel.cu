#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void unique_idx_calc_threadIdx(int * input)
{
	int tid = threadIdx.x;
	printf("threadIdx: %d, value: %d \n", tid, input[tid]);
}

int main()
{
	int array_size = 8;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33 };

	for (size_t i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}
	printf("\n \n");
	
	int * d_data;
	hipMalloc((void**) &d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(1);

	unique_idx_calc_threadIdx << < grid, block >> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
    return 0;
}
