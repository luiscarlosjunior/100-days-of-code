#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

void printThreadIds();

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void hello_cuda()
{
	printf("Hello Cuda world \n");
}

__global__ void print_threadIds()
{
	printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d\n",
		threadIdx.x, threadIdx.y, threadIdx.z);
}

// Print blocks and grid
__global__ void print_threadIds()
{
	printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d blockDim.x: %d, blockDim.y: %d\n",
		threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	// HelloWorld();

	printThreadIds();

	return 0;
}

void printThreadIds() {
	int nx, ny;
	nx = 16;
	ny = 16;

	dim3 block(8, 8);
	dim3 grid(nx / block.x, ny / block.y);

	print_threadIds << < grid, block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();

}

void HelloWorld() {
	// Executa CUDA World

	int nx, ny;
	nx = 16;
	ny = 4;


	dim3 block(8, 2);
	dim3 grid(nx / block.x, 2);

	hello_cuda << < grid, block >> > ();
	hipDeviceSynchronize();
	
}
